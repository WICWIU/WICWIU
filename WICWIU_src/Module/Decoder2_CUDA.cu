
#include <hip/hip_runtime.h>
#ifdef __CUDNN__



// #include "../LossFunction/SoftmaxCrossEntropy.hpp"
//
// // template class SoftmaxCrossEntropy<int>;
// template class SoftmaxCrossEntropy<float>;

// #include "Decoder2.hpp"
#include "LinearLayer.hpp"
#include "EmbeddingLayer.hpp"
#include "RecurrentLayer.hpp"
#include "LSTM2Layer.hpp"
#include "GRULayer.hpp"
#include "Decoder2.hpp"


// template class LRelu<int>;
template class Decoder2<float>;

/*!
@class LRelu cuda
*/
__global__ void ForwardPropagate_kernel(float *pDevEncoderHidden, float *pDevinitHidden, int batchIndex, int colSize) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < colSize; idx += blockDim.x * gridDim.x) {

          int startIndex = colSize * batchIndex;

          pDevinitHidden[startIndex + idx] = pDevEncoderHidden[startIndex + idx];

    }
}
/*!
  EncoderLengths가 NULL일때는 고려하지 않고 구현!!!
  추후 추가 필요!
*/

template<typename DTYPE> int Decoder2<DTYPE>::ForwardPropagateOnGPU(int pTime) {

      // std::cout<<"Decoder2<DTYPE>::ForwardPropagateOnGPU "<<pTime<<'\n';

  //encoder에서 decoder로 복사!
      if(pTime == 0){

          int noBlock = 3, threadsPerBlock = 128;

          Tensor<DTYPE> *encoderLengths = m_EncoderLengths->GetResult();

          //Data 접근!
          Tensor<DTYPE> *_initHidden = this->GetInput()[1]->GetResult();
          Tensor<DTYPE> *initHidden = m_initHiddenTensorholder->GetResult();

          // std::cout<<"Encoder last hidden value"<<'\n';
          // std::cout<<_initHidden->GetShape()<<'\n';
          // std::cout<<_initHidden<<'\n';

          //batchsize, colsize
          int batchsize  = _initHidden->GetBatchSize();
          int colSize    = _initHidden->GetColSize();

          // std::cout<<"복사 전"<<'\n';
          // std::cout<<initHidden->GetShape()<<'\n';
          // std::cout<<initHidden<<'\n';

          // std::cout<<"batch size : "<<batchsize<<'\n';

          for(int ba = 0; ba < batchsize; ba++){

              DTYPE *m_pDevEncoderHidden  = _initHidden->GetGPUData((*encoderLengths)[ba]-1);
              DTYPE *m_pDevinitHidden  = initHidden->GetGPUData(0);

              ForwardPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevEncoderHidden, m_pDevinitHidden, ba, colSize);

          }

          // std::cout<<"복사 해온 값"<<'\n';
          // std::cout<<initHidden->GetShape()<<'\n';
          // std::cout<<initHidden<<'\n';
          // //
          // std::cout<<"Encoder length"<<'\n';
          // std::cout<<encoderLengths<<'\n';

      }

      int numOfExcutableOperator = this->GetNumOfExcutableOperator();
      Container<Operator<DTYPE> *> *ExcutableOperator = this->GetExcutableOperatorContainer();

      for (int i = 0; i < numOfExcutableOperator; i++) {
          (*ExcutableOperator)[i]->ForwardPropagateOnGPU(pTime);
      }
      return TRUE;
}

/*!
@brief LRelu의 BackPropagate 커널함수
@details BackPropagateOnGPU에서 호출되어 실행
@see int LRelu<DTYPE>::BackPropagateOnGPU(int pTime = 0)
@details 1차원으로 배열 된 block과 thread에 접근하여 연산
@param pDevOutput LRelu ForwardPropagate연산의 결과인 output값의 GPU data
@param pDevDelta LRelu 다음 Operator의 BackPropagate 결과 값인 delta의 GPU data.
@param pDevInputDelta 연산의 결과인 delta값을 저장할 GPU data.
@param negativeSlope output값이 0.f 이하일 때 사용하는 기울기값
@param weightDim LRelu연산의 결과값의 dimension.
*/
__global__ void BackPropagate_kernel(float *pDevEncoderHidden, float *pDevDecoderHidden, int batchIndex, int colSize) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < colSize; idx += blockDim.x * gridDim.x) {

          int startIndex = colSize * batchIndex;

          pDevEncoderHidden[startIndex + idx] = pDevDecoderHidden[startIndex + idx];

    }
}


template<typename DTYPE> int Decoder2<DTYPE>::BackPropagateOnGPU(int pTime) {

      //seq2seq에 맞춰서 수정하기!!! --> Decoder안에 embedding, rnn, linear 다 있어서 layer별로 time을 다 실행하고 나서 넘어가야 됨!
      if(pTime !=0)
        return TRUE;
      //그래서 딱 한번만 호출되고 안쪽에서 모든 time에 대해 처리하도록!

      int numOfExcutableOperator = this->GetNumOfExcutableOperator();
      Container<Operator<DTYPE> *> *ExcutableOperator = this->GetExcutableOperatorContainer();

      // std::cout<<"backward 호출 전 initHidden gradient 값"<<'\n';
      // std::cout<<m_initHiddenTensorholder->GetGradient()<<'\n';

      // for (int i = numOfExcutableOperator - 1; i >= 0; i--) {
      //     (*ExcutableOperator)[i]->BackPropagateOnGPU(pTime);                   // 여기 어차피 한번만 돌려버리면 되는거 아닌가  --> embedding도 있어서 안됨!
      // }

      //seq2seq에 맞춰서 수정하기!!! --> Decoder안에 embedding, rnn, linear 다 있어서 layer별로 time을 다 실행하고 나서 넘어가야 됨!
      for (int i = numOfExcutableOperator - 1; i >= 0; i--) {
          for(int ti = timesize-1; ti>=0; ti--){
            (*ExcutableOperator)[i]->BackPropagateOnGPU(ti);
        }
      }


      //decoder에서 encoder로 복사!
      if(pTime == 0){

          int noBlock = 3, threadsPerBlock = 128;

          Tensor<DTYPE> *encoderLengths = m_EncoderLengths->GetResult();

          //Data 접근!
          Tensor<DTYPE> *_initHidden = this->GetInput()[1]->GetGradient();
          Tensor<DTYPE> *initHidden = m_initHiddenTensorholder->GetGradient();

          //batchsize, colsize
          int batchsize  = _initHidden->GetBatchSize();
          int colSize    = _initHidden->GetColSize();

          // std::cout<<"복사 전 encoder의 gradient"<<'\n';
          // std::cout<<_initHidden<<'\n';

          // std::cout<<"initHidden의 gradient"<<'\n';
          // std::cout<<initHidden<<'\n';

          for(int ba = 0; ba < batchsize; ba++){

              DTYPE *m_pDevEncoderHidden  = _initHidden->GetGPUData((*encoderLengths)[ba]-1);
              DTYPE *m_pDevinitHidden  = initHidden->GetGPUData(0);

              BackPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevEncoderHidden, m_pDevinitHidden, ba, colSize);

          }

          // std::cout<<"복사 후 encoder의 gradient"<<'\n';
          // std::cout<<_initHidden<<'\n';
          //
          //
          // std::cout<<"Encoder length"<<'\n';
          // std::cout<<encoderLengths<<'\n';

      }

      return TRUE;
}

#endif  // ifdef __CUDNN__
